#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include "helper_cuda_gl.h"
#include "helper_functions.h"
#include "hip/hip_vector_types.h"
#include "helper_string.h"
#include "helper_timer.h"
#include "hipblas.h"
#include "src/training/training.cu"
#include "src/testing/testing.cu"
#include "src/common/cuTimer.cu"
#include "src/common/parseinputs.cpp"


void runmulticlassifier(char* ,int ,int ,char* ,int ,int ,int ,float ,float ,int ,float ,float, float, float);

//MultiClass classification using SVM
int main(int argc, char** argv)
{

    int lineNum = 0;
    FILE *fp = NULL;
    fp = fopen("conf.txt","r");

    char strLine[256];
    fgets(strLine,256,fp);
    lineNum=atoi(strLine);

    const char *split = " ";
    char *trainFileName;
    char *testFileName;
    char *featureNum;
    char *trainingSampleNum;
    char *testSampleNum;
    char *classNum;
    char *C;
    char *gamma;	

    for (int i=0; i < lineNum; i++){
        fgets(strLine,256,fp);

        trainFileName=strtok(strLine,split);

        testFileName=strtok(NULL,split);

        featureNum=strtok(NULL,split);

        trainingSampleNum=strtok(NULL,split);

        testSampleNum=strtok(NULL,split);

        classNum=strtok(NULL,split);

        C=strtok(NULL,split);
        gamma=strtok(NULL,split);

        runmulticlassifier(trainFileName,
                atoi(trainingSampleNum),
                atoi(featureNum) ,
                testFileName,
                atoi(testSampleNum),
                1,
                atoi(classNum),
                atoi(C),
                0.001,
                0,
                atof(gamma),
                0,
                0,
                0);
    }

}

/**
 * Runs both training and testing. Provides timings
 * @param trainfilename name of the file containing the training samples
 * @param ntraining number of training samples
 * @param nfeatures number of features in the each training sample
 * @param testfilename name of the file containing the testing samples
 * @param ntesting number of testing samples
 * @param code {0: One vs All, 1: All vs All, 2: Even vs Odd}
 * @param nclasses number of classes in the SVM problem
 * @param ntasks number of binary classification tasks
 * @param C penalization parameter
 * @param tau stopping parameter of the SMO algorithm
 * @param kernelcode type of kernel to use {0 (RBF), 1(linear), 2(polynomial), 3(sigmoid)}
 * @param gamma if using RBF kernel, the value of gamma
 * @param a if using polynomial or sigmoid kernel
 * @param b if using polynomial or sigmoid kernel
 * @param d if using polynomial kernel
 */
void runmulticlassifier(char* trainfilename,
                        int ntraining,
                        int nfeatures,
                        char* testfilename,
                        int ntesting,
                        int code,
                        int nclasses,
                        float C,
                        float tau,
                        int kernelcode,
                        float gamma,
                        float a,
                        float b,
                        float d)
{
    
    for(float gamma=0.1; gamma <100.0 ; gamma*=3 ){
	    hipblasStatus_t status;

	    status = cublasInit();
	    if (status != HIPBLAS_STATUS_SUCCESS) {
	        fprintf (stderr, "!!!! CUBLAS initialization error\n");
	    }
	    
	    // Set-Up

		//Allocate memory for xtraindata
		float* h_xtraindata = (float*) malloc(sizeof(float) * ntraining* nfeatures);
		float* h_xtraindatatemp = (float*) malloc(sizeof(float) * ntraining* nfeatures);

		//Allocate memory for xtestdata
		float* h_xtestdata = (float*) malloc(sizeof(float) * ntesting* nfeatures);
		float* h_xtestdatatemp = (float*) malloc(sizeof(float) * ntesting* nfeatures);


		//Allocate memory for traindata_label
		int* h_ytraindata = (int*) malloc(sizeof(int) * ntraining);

		//Allocate memory for testdata_label
		int* h_ytestdata = (int*) malloc(sizeof(int) * ntesting);
		int* h_rdata;
	    
	    memset(h_xtraindata, 0, sizeof(float) * ntraining* nfeatures);
	    memset(h_xtraindatatemp, 0, sizeof(float) * ntraining* nfeatures);
	    memset(h_xtestdata, 0, sizeof(float) * ntesting* nfeatures);
	    memset(h_xtestdatatemp, 0, sizeof(float) * ntesting* nfeatures);

	    memset(h_ytraindata, 0, sizeof(int) * ntraining);
	    memset(h_ytestdata, 0, sizeof(int) * ntesting);
	    
		//Parse data from input file
		printf("Parsing input data...\n");
		parsedatalibsvm(trainfilename, h_xtraindatatemp, h_ytraindata, ntraining, nfeatures, nclasses);
		parsedatalibsvm(testfilename, h_xtestdatatemp, h_ytestdata, ntesting, nfeatures, nclasses);
		//printdata(h_xtestdatatemp, h_ytestdata,  ntesting, nfeatures);
		printf("Parsing input data done!\n");

		int ntasks;

		if( code==0 )
		{
			printf("Code: One Vs All\n");
			ntasks = nclasses;
			//Allocate memory for rdata
		    h_rdata= (int*) malloc(sizeof(int) * nclasses * ntasks);
			generateovacode(h_rdata, nclasses, ntasks);
		}
		else if( code==1 )
		{
			printf("Code All vs All\n");
			ntasks = nclasses*(nclasses-1)/2;
			//Allocate memory for rdata
		    h_rdata= (int*) malloc(sizeof(int) * nclasses * ntasks);
			generateavacode(h_rdata, nclasses, ntasks);
		}
		else if( code==2 )
		{
			printf("Code Odd vs Even\n");
			//Allocate memory for rdata
			ntasks=2;
		    h_rdata= (int*) malloc(sizeof(int) * nclasses * ntasks);
			generateevenoddcode(h_rdata, nclasses, ntasks);
		}

		printcode(h_rdata, nclasses, ntasks);

		float* h_C = (float*) malloc(sizeof(float) * ntasks);
	    for(int i=0; i<ntasks; i++)
	    {
	        h_C[i]=C;
	    }

	    printf("Input Train File Name: %s\n", trainfilename);
	    printf("Input Test File Name: %s\n", testfilename);

		printf("# of training samples: %i\n", ntraining);
		printf("# of testing samples: %i\n", ntesting);
		printf("# of features: %i\n", nfeatures);
		printf("# of tasks: %i\n", ntasks);
		printf("# of classes: %i\n", nclasses);
		printf("Gamma: %f\n", gamma);

		bool iszero=false;

		for (int i=0; i< ntraining; i++)
		{
			for (int j=0; j<nfeatures; j++)
			{   
				//h_xtraindata is the transpose matrix  of h_xtraindatatemp
				h_xtraindata[j*ntraining+i]=h_xtraindatatemp[i*nfeatures+j];
			}
			if(h_ytraindata[i]==0)
			{
				iszero=true;
			}
		}

		for (int i=0; i< ntesting; i++)
		{
			for (int j=0; j<nfeatures; j++)
			{
				//h_xtestdata is the transpose matrix of h_xtestdatatemp
				h_xtestdata[j*ntesting+i]=h_xtestdatatemp[i*nfeatures+j];
			}
		}

		if (iszero)
		{
			for (int i=0; i< ntraining; i++)
			{
				h_ytraindata[i]=h_ytraindata[i]+1;
			}
			for (int i=0; i< ntesting; i++)
			{
				h_ytestdata[i]=h_ytestdata[i]+1;
			}
		}

		free(h_xtraindatatemp);
		free(h_xtestdatatemp);

		int* h_ltesthatdata = (int*) malloc(sizeof(int) * ntesting);

		//Allocate memory for b
		float * h_b= (float*) malloc(sizeof(float) * ntasks);
		for (int i=0; i<ntasks; i++)
		{
			h_b[i]= 0.0f;
		}

		//Allocate memory for adata
		float* h_atraindata= (float*) malloc(sizeof(int) * ntraining * ntasks);

		cuResetTimer();
		float tA1=cuGetTimer();
		
		printf("Training classifier...\n");
		trainclassifier(h_xtraindata,
						h_ytraindata,
						h_rdata,
						h_atraindata,
						ntraining,
						nfeatures,
						nclasses,
						ntasks,
						h_C,
						h_b,
						tau,
						kernelcode,
					    gamma,
						a,
						b,
						d);

		float tA2=cuGetTimer();

		printf("Training classifier done!\n");
		printf("Training time Launch =%.1f usec, finished=%.1f msec\n",tA1*1.e3,tA2);
	    
		for (int j=0; j<ntasks; j++)
		{
			int svnum=0;
			for (int i=0; i<ntraining; i++)
			{
				if(h_atraindata[j*ntraining + i]!=0)
				{
					svnum++;
				}
			}
			printf("Task %i, svnum, %i, b %f\n",j, svnum,h_b[j] );
		}

		int nSV=0;
		for (int i=0; i< ntraining; i++)
		{
			for (int j=0; j< ntasks; j++)
			{
				if(h_atraindata[j*ntraining+i]!=0)
				{
					nSV++;
					break;
				}
			}
		}

		float* h_xtraindatared = (float*) malloc(sizeof(float) * nSV* nfeatures);
		int* h_ltraindatared = (int*) malloc(sizeof(int) * nSV);
		float* h_atraindatared = (float*) malloc(sizeof(float) *ntasks* nSV);

		int k=0;

		for (int i=0; i< ntraining; i++)
		{
			//Check if SV in any tasks
			bool isSV=false;

			for (int j=0; j< ntasks; j++)
			{
				if(h_atraindata[j*ntraining+i]!=0)
				{
					isSV=true;
					break;
				}
			}

			//If SV then copy sample and alphas
			if(isSV)
			{
				for (int j=0; j< ntasks; j++)
				{
					h_atraindatared[j*nSV +k]= h_atraindata[j*ntraining+i];
				}


				for (int j=0; j<nfeatures; j++)
				{
					h_xtraindatared[j*nSV+k]=h_xtraindata[j*ntraining+i];
				}
				h_ltraindatared[k]= h_ytraindata[i];

				k++;
			}
		}


		printf("Testing classifier...\n");

		cuResetTimer();
		float tB1=cuGetTimer();
		testingclassifier(	h_xtraindatared,
							h_xtestdata,
							h_ltraindatared,
							h_ltesthatdata,
							h_rdata,
							h_atraindatared,
							nSV,
							ntesting,
							nfeatures,
							nclasses,
							ntasks,
							h_b,
							gamma,
							a,
							b,
							d,
							kernelcode);

		printf("Testing classifier done\n");
		float tB2=cuGetTimer();
		printf("Testing time Launch =%.1f usec, finished=%.1f msec\n",tB1*1.e3,tB2);

		int errors=0;

		for (int i=0; i<ntesting; i++)
		{
			if( h_ytestdata[i]!=h_ltesthatdata[i])
			{
				errors++;
			}
		}


		printf("%f # of testing samples %i, # errors %i, Rate %f\n",gamma, ntesting, errors, 100* (float) (ntesting -errors)/(float)ntesting);

		free(h_rdata);
		free(h_xtraindata);
		free(h_xtestdata);
		free(h_ytraindata);
		free(h_ytestdata);
		free(h_b);
		free(h_atraindata);
		free(h_xtraindatared);
		free(h_ltraindatared);
		free(h_atraindatared);
    }
} 

