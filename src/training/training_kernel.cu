#include "hip/hip_runtime.h"

#ifndef _TRAINING_KERNEL_H_
#define _TRAINING_KERNEL_H_

#include <stdio.h>

/**
 * Set initial values of the binary labels and alphas
 * @param d_ltraindata device pointer to multiclass labels
 * @param d_rdata device pointer to the binary matrix that encodes the output code
 * @param d_ytraindata device pointer to the array with binary labels
 * @param d_atraindata device pointer to the array with the alphas
 * @param d_fdata device pointer to the intermediate values of f
 * @param ntraining number of training samples in the training set
 * @param ntasks number of binary tasks to be solved
 * @param d_active device pointer to the binary array that indicates the status of the task
 */
template <unsigned int blockSize, bool isNtrainingPow2>
__global__ static void initializetraining(	int* d_ltraindata,
											int* d_rdata,
											int* d_ytraindata,
											float* d_atraindata,
											float* d_fdata,
											int ntraining,
											int ntasks,
											int* d_active)
{
	const unsigned int j = blockIdx.y;
	unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
	const unsigned int gridSize = blockSize*2*gridDim.x;

	const unsigned int bidy= d_active[j];

	while (i < ntraining)
	{
		int label= d_ltraindata[i];
		d_ytraindata[bidy*ntraining + i]= d_rdata[(label-1)*ntasks + bidy];
		d_atraindata[bidy* ntraining + i]=0.0f;
		d_fdata[bidy* ntraining + i]= -1.0* (float)( d_ytraindata[bidy*ntraining + i]);

		if (isNtrainingPow2 || i + blockSize < ntraining)
		{
			label= d_ltraindata[i + blockSize];
			d_ytraindata[bidy*ntraining + i + blockSize]= d_rdata[(label-1)*ntasks + bidy];
			d_atraindata[bidy* ntraining + i + blockSize]=0.0f;
			d_fdata[bidy* ntraining + i + blockSize]= -1.0* (float)( d_ytraindata[bidy*ntraining + i + blockSize]);

		}
		i += gridSize;
	}
	__syncthreads();

}

/**
 * Calculate the new values of the chosen duple of alphas
 * @param d_xtraindata device pointer to the training set
 * @param d_kdata device pointer to the cached rows of the gram matrix
 * @param d_ytraindata device pointer to the array with binary labels
 * @param d_atraindata device pointer to the array with the alphas
 * @param d_anewtraindata device pointer to the new duple of alphas (alpha 1 and alpha 2)
 * @param d_aoldtraindata device pointer to the old duple of alphas (alpha 1 and alpha 2)
 * @param d_fdata device pointer to the intermediate values of f
 * @param d_Iup_global device pointer to the Iup indexes for each binary task
 * @param d_Ilow_global device pointer to the Ilow indexes for each binary task
 * @param d_Iup_cache device pointer to the location of the Iup index in the kernel cache
 * @param d_Ilow_cache device pointer to the location of the Ilow index in the kernel cache
 * @param d_done device pointer containing the status of each binary task
 * @param ntraining number of training samples in the training set
 * @param nfeatures number of features in each of the training samples
 * @param ntasks number of binary tasks to be solved
 * @param d_C device pointer to the regularization parameter for each binary task
 */
__global__ static void calculatealphas(	float* d_xtraindata,
										float* d_kdata,
										int* d_ytraindata,
										float* d_atraindata,
										float* d_anewtraindata,
										float* d_aoldtraindata,
										float* d_fdata,
										int* d_Iup_global,
										int* d_Ilow_global,
										int* d_Iup_cache,
										int* d_Ilow_cache,
										int* d_done,
										int ntraining,
										int nfeatures,
										int ntasks,
										float* d_C)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int bidx = blockIdx.x;
	const float eps= 0.000001;

	//Check if the task has converged
	if(d_done[tid]==0)
	{

		int blockYAlpha=(int)ceil((float)(ntasks)/(float)(TPB));

		if((bidx != blockYAlpha-1) || ((bidx == blockYAlpha-1) && tid < (ntasks - bidx*TPB)))
		{

			float C= d_C[tid];

			int g_Iup=d_Iup_global[tid];
			int g_Ilow=d_Ilow_global[tid];

			int y_up= d_ytraindata[tid*ntraining + g_Iup];
			int y_low= d_ytraindata[tid*ntraining + g_Ilow];

			float alpha_up_old = d_atraindata[tid*ntraining + g_Iup];
			float alpha_low_old = d_atraindata[tid*ntraining + g_Ilow];
			float alpha_up_new =0.0;
			float alpha_low_new =0.0;

			d_aoldtraindata[tid*2]= alpha_low_old;
			d_aoldtraindata[tid*2 +1]= alpha_up_old;


			float f_up_old= d_fdata[tid*ntraining + g_Iup];
			float f_low_old= d_fdata[tid*ntraining + g_Ilow];

			int s= y_up * y_low;


			float gamma=0.0f;
			float L=0.0f;
			float H=0.0f;

			// get L and H in the range (0,C)

			if(y_up == y_low)
			{
				gamma= alpha_low_old + alpha_up_old;
			}
			else
			{
				gamma= alpha_low_old - alpha_up_old;
			}

			if(s==1) // y_up == y_low =1 or -1 
			{
				L= max( 0.0f, gamma-C);
				H= min (C,gamma);
			}
			else
			{
				L= max(0.0f, -gamma);
				H= min(C, C-gamma);
			}


			if(H<=L)
			{
				d_done[tid]=1;

			}
            

			float K12= d_kdata[d_Ilow_cache[tid]*ntraining + g_Iup];
			float K11= d_kdata[d_Ilow_cache[tid]*ntraining + g_Ilow];
			float K22= d_kdata[d_Iup_cache[tid]*ntraining + g_Iup];

			float nu= 2*K12 - K11 -K22;
	
			if(nu < 0)
			{
				alpha_up_new= alpha_up_old - (y_up*(f_low_old - f_up_old)/nu);
				if(alpha_up_new <L)
				{
					alpha_up_new=L;
				}
				else if (alpha_up_new>H)
				{
					alpha_up_new=H;
				}
			}
			else
			{
				float slope= y_up *(f_low_old - f_up_old);
				float change= slope * (H-L);
				if(fabs(change)>0.0f)
				{
					if(slope>0.0f)
					{
						alpha_up_new= H;
					}
					else
					{
						alpha_up_new= L;
					}
				}
				else
				{
					alpha_up_new= alpha_up_old;
				}

				if( alpha_up_new > C - eps * C)
				{
					alpha_up_new=C;
				}

				else if (alpha_up_new < eps * C)
				{
					alpha_up_new=0.0f;
				}
			}

			if( fabs( alpha_up_new - alpha_up_old) < eps * ( alpha_up_new + alpha_up_old + eps))
			{
				d_done[tid]=1;
			}

			if(s==1)
			{
				alpha_low_new= gamma - alpha_up_new;
			}
			else
			{
				alpha_low_new= gamma + alpha_up_new;
			}

			if( alpha_low_new > C - eps * C)
			{
				alpha_low_new = C;
			}
			else if (alpha_low_new < eps * C)
			{
				alpha_low_new = 0.0f;
			}

			d_anewtraindata[tid*2] = alpha_low_new;
			d_anewtraindata[tid*2+1] = alpha_up_new;

		}

	}
	__syncthreads();
}

/**
 * Calculate the new values of every training sample's f
 * @param d_xtraindata device pointer to the training set
 * @param d_kdata device pointer to the cached rows of the gram matrix
 * @param d_ytraindata device pointer to the array with binary labels
 * @param d_atraindata device pointer to the array with the alphas
 * @param d_anewtraindata device pointer to the new duple of alphas (alpha 1 and alpha 2)
 * @param d_aoldtraindata device pointer to the old duple of alphas (alpha 1 and alpha 2)
 * @param d_fdata device pointer to the intermediate values of f
 * @param d_Iup_global device pointer to the Iup indexes for each binary task
 * @param d_Ilow_global device pointer to the Ilow indexes for each binary task
 * @param d_Iup_cache device pointer to the location of the Iup index in the kernel cache
 * @param d_Ilow_cache device pointer to the location of the Ilow index in the kernel cache
 * @param d_done device pointer containing the status of each binary task
 * @param ntraining number of training samples in the training set
 * @param nfeatures number of features in each of the training samples
 * @param activeTasks number of non converged tasks
 * @param ntasks number of binary tasks to be solved
 * @param d_C device pointer to the regularization parameter for each binary task
 */
template <unsigned int blockSize, bool isNtrainingPow2>
__global__ static void updateparams(float* d_xtraindata,
									float* d_kdata,
									int* d_ytraindata,
									float* d_atraindata,
									float* d_anewtraindata,
									float* d_aoldtraindata,
									float* d_fdata,
									int* d_Iup_global,
									int* d_Ilow_global,
									int* d_Iup_cache,
									int* d_Ilow_cache,
									int* d_done,
									int* d_active,
									int ntraining,
									int nfeatures,
									int ntasks,
									int activeTasks,
									float* d_C)
{

	const unsigned int tid = threadIdx.x;
	const unsigned int bidx = blockIdx.x;
	unsigned int j = blockIdx.y;
	unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;


	int bidy= d_active[j];

	if(d_done[bidy]==0)
	{
		int g_Iup=d_Iup_global[bidy];
		int g_Ilow=d_Ilow_global[bidy];

		float alpha_low_new= d_anewtraindata[bidy*2];
		float alpha_up_new= d_anewtraindata[bidy*2+1];

		float alpha_low_old= d_aoldtraindata[bidy*2];
		float alpha_up_old= d_aoldtraindata[bidy*2 +1];

		int y_up= d_ytraindata[bidy*ntraining + g_Iup];
		int y_low= d_ytraindata[bidy*ntraining + g_Ilow];

		while (i < ntraining)
		{

			float f_i_old = d_fdata[bidy* ntraining+i];

			float Klowi = d_kdata[d_Ilow_cache[bidy] * ntraining +i];
			float Kupi = d_kdata[d_Iup_cache[bidy] * ntraining + i];

			float f_i_new = f_i_old + (alpha_low_new - alpha_low_old)*y_low* Klowi + (alpha_up_new - alpha_up_old)*y_up* Kupi;

			d_fdata[bidy*ntraining + i] = f_i_new;


			if (isNtrainingPow2 || i + blockSize < ntraining)
			{
				f_i_old = d_fdata[bidy*ntraining +i + blockSize];
				Klowi = d_kdata[d_Ilow_cache[bidy] * ntraining + i + blockSize];
				Kupi = d_kdata[d_Iup_cache[bidy] * ntraining + i + blockSize];

				f_i_new= f_i_old + (alpha_low_new - alpha_low_old)*y_low* Klowi + (alpha_up_new - alpha_up_old)*y_up*Kupi;

				d_fdata[bidy*ntraining + i + blockSize]= f_i_new;
			}

			i += gridSize;
		}

		if(bidx==0 && tid==0)
		{
			d_atraindata[bidy*ntraining + g_Ilow]= d_anewtraindata[bidy*2];
			d_atraindata[bidy*ntraining + g_Iup]= d_anewtraindata[bidy*2+1];
		}
	}
}
/**
 * Calculate the new values of every training sample's f and alphaIup and alphaIlow
 * @param d_xtraindata device pointer to the training set
 * @param d_kdata device pointer to the cached rows of the gram matrix
 * @param d_ytraindata device pointer to the array with binary labels
 * @param d_atraindata device pointer to the array with the alphas
 * @param d_anewtraindata device pointer to the new duple of alphas (alpha 1 and alpha 2)
 * @param d_aoldtraindata device pointer to the old duple of alphas (alpha 1 and alpha 2)
 * @param d_fdata device pointer to the intermediate values of f
 * @param d_Iup_global device pointer to the Iup indexes for each binary task
 * @param d_Ilow_global device pointer to the Ilow indexes for each binary task
 * @param d_Iup_cache device pointer to the location of the Iup index in the kernel cache
 * @param d_Ilow_cache device pointer to the location of the Ilow index in the kernel cache
 * @param d_done device pointer containing the status of each binary task
 * @param ntraining number of training samples in the training set
 * @param nfeatures number of features in each of the training samples
 * @param activeTasks number of non converged tasks
 * @param ntasks number of binary tasks to be solved
 * @param d_C device pointer to the regularization parameter for each binary task
 */
template <unsigned int blockSize, bool isNtrainingPow2>
__global__ static void merge2kernel( float* d_xtraindata,
										    float* d_kdata,
											int* d_ytraindata,
											float* d_atraindata,
											float* d_anewtraindata,
											float* d_aoldtraindata,
											float* d_fdata,
											int* d_Iup_global,
											int* d_Ilow_global,
											int* d_Iup_cache,
											int* d_Ilow_cache,
											int* d_done,
											int* d_active,
											int ntraining,
											int nfeatures,
											int ntasks,
											int activeTasks,
											float* d_C)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int bidx = blockIdx.x;
	const float eps= 0.000001;

	//Check if the task has converged
	if(d_done[tid]==0)
	{

		int blockYAlpha=(int)ceil((float)(ntasks)/(float)(TPB));

		if((bidx != blockYAlpha-1) || ((bidx == blockYAlpha-1) && tid < (ntasks - bidx*TPB)))
		{

			float C= d_C[tid];

			int g_Iup=d_Iup_global[tid];
			int g_Ilow=d_Ilow_global[tid];

			int y_up= d_ytraindata[tid*ntraining + g_Iup];
			int y_low= d_ytraindata[tid*ntraining + g_Ilow];

			float alpha_up_old = d_atraindata[tid*ntraining + g_Iup];
			float alpha_low_old = d_atraindata[tid*ntraining + g_Ilow];
			float alpha_up_new =0.0;
			float alpha_low_new =0.0;

			d_aoldtraindata[tid*2]= alpha_low_old;
			d_aoldtraindata[tid*2 +1]= alpha_up_old;


			float f_up_old= d_fdata[tid*ntraining + g_Iup];
			float f_low_old= d_fdata[tid*ntraining + g_Ilow];

			int s= y_up * y_low;


			float gamma=0.0f;
			float L=0.0f;
			float H=0.0f;

			// get L and H in the range (0,C)

			if(y_up == y_low)
			{
				gamma= alpha_low_old + alpha_up_old;
			}
			else
			{
				gamma= alpha_low_old - alpha_up_old;
			}

			if(s==1) // y_up == y_low =1 or -1 
			{
				L= max( 0.0f, gamma-C);
				H= min (C,gamma);
			}
			else
			{
				L= max(0.0f, -gamma);
				H= min(C, C-gamma);
			}


			if(H<=L)
			{
				d_done[tid]=1;

			}
            

			float K12= d_kdata[d_Ilow_cache[tid]*ntraining + g_Iup];
			float K11= d_kdata[d_Ilow_cache[tid]*ntraining + g_Ilow];
			float K22= d_kdata[d_Iup_cache[tid]*ntraining + g_Iup];

			float nu= 2*K12 - K11 -K22;
	
			if(nu < 0)
			{
				alpha_up_new= alpha_up_old - (y_up*(f_low_old - f_up_old)/nu);
				if(alpha_up_new <L)
				{
					alpha_up_new=L;
				}
				else if (alpha_up_new>H)
				{
					alpha_up_new=H;
				}
			}
			else
			{
				float slope= y_up *(f_low_old - f_up_old);
				float change= slope * (H-L);
				if(fabs(change)>0.0f)
				{
					if(slope>0.0f)
					{
						alpha_up_new= H;
					}
					else
					{
						alpha_up_new= L;
					}
				}
				else
				{
					alpha_up_new= alpha_up_old;
				}

				if( alpha_up_new > C - eps * C)
				{
					alpha_up_new=C;
				}

				else if (alpha_up_new < eps * C)
				{
					alpha_up_new=0.0f;
				}
			}

			if( fabs( alpha_up_new - alpha_up_old) < eps * ( alpha_up_new + alpha_up_old + eps))
			{
				d_done[tid]=1;
			}

			if(s==1)
			{
				alpha_low_new= gamma - alpha_up_new;
			}
			else
			{
				alpha_low_new= gamma + alpha_up_new;
			}

			if( alpha_low_new > C - eps * C)
			{
				alpha_low_new = C;
			}
			else if (alpha_low_new < eps * C)
			{
				alpha_low_new = 0.0f;
			}

			d_anewtraindata[tid*2] = alpha_low_new;
			d_anewtraindata[tid*2+1] = alpha_up_new;

		}

	}
	__syncthreads();

	unsigned int j = blockIdx.y;
	unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;


	int bidy= d_active[j];

	if(d_done[bidy]==0)
	{
		int g_Iup=d_Iup_global[bidy];
		int g_Ilow=d_Ilow_global[bidy];

		float alpha_low_new= d_anewtraindata[bidy*2];
		float alpha_up_new= d_anewtraindata[bidy*2+1];

		float alpha_low_old= d_aoldtraindata[bidy*2];
		float alpha_up_old= d_aoldtraindata[bidy*2 +1];

		int y_up= d_ytraindata[bidy*ntraining + g_Iup];
		int y_low= d_ytraindata[bidy*ntraining + g_Ilow];

		while (i < ntraining)
		{

			float f_i_old = d_fdata[bidy* ntraining+i];

			float Klowi = d_kdata[d_Ilow_cache[bidy] * ntraining +i];
			float Kupi = d_kdata[d_Iup_cache[bidy] * ntraining + i];

			float f_i_new = f_i_old + (alpha_low_new - alpha_low_old)*y_low* Klowi + (alpha_up_new - alpha_up_old)*y_up* Kupi;

			d_fdata[bidy*ntraining + i] = f_i_new;


			if (isNtrainingPow2 || i + blockSize < ntraining)
			{
				f_i_old = d_fdata[bidy*ntraining +i + blockSize];
				Klowi = d_kdata[d_Ilow_cache[bidy] * ntraining + i + blockSize];
				Kupi = d_kdata[d_Iup_cache[bidy] * ntraining + i + blockSize];

				f_i_new= f_i_old + (alpha_low_new - alpha_low_old)*y_low* Klowi + (alpha_up_new - alpha_up_old)*y_up*Kupi;

				d_fdata[bidy*ntraining + i + blockSize]= f_i_new;
			}

			i += gridSize;
		}

		if(bidx==0 && tid==0)
		{
			d_atraindata[bidy*ntraining + g_Ilow]= d_anewtraindata[bidy*2];
			d_atraindata[bidy*ntraining + g_Iup]= d_anewtraindata[bidy*2+1];
		}
	}



}


#endif // _TRAINING_KERNEL_H_
